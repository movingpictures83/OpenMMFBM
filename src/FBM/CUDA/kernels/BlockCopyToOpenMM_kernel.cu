
#include <hip/hip_runtime.h>
__global__ void blockcopyToOpenMM( float *target, float *source, int *blocks, int numblocks, int setnum, int N ) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
    int dof = 3 * blocks[blockNum] + setnum;
    int atom = dof / 3;

    if( atom >= N || ( blockNum != numblocks && atom >= blocks[blockNum + 1] ) ) {
        return;    // Out of bounds
    }

    *( target + ( dof + atom + 1 )*sizeof( float ) ) = source[dof]; // Save the old
}
